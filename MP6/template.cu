#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix AuxSum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

  __global__ void add(float *input, float *output, int len) {
    int idx1 = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    int idx2 = idx1 + blockDim.x;

    int input_value = (blockIdx.x != 0) ? input[blockIdx.x - 1] : 0;

    output[idx1] += input_value * (idx1 < len);
    output[idx2] += input_value * (idx2 < len);
}

__global__ void scan(float *input, float *output, int len, float *AuxSum) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host
 __shared__ float T[2*BLOCK_SIZE];
  int t = threadIdx.x;
  int start = 2 * blockIdx.x * BLOCK_SIZE;

  T[t] = (start + t < len) ? input[start + t] : 0;
  T[BLOCK_SIZE + t] = (start + BLOCK_SIZE + t < len) ? input[start + BLOCK_SIZE + t] : 0;


  int stride = 1;
  while(stride < 2*BLOCK_SIZE) {
    __syncthreads();
    int index = (t+1)*stride*2 - 1;
    if(index < 2*BLOCK_SIZE && (index-stride) >= 0)
      T[index] += T[index-stride];
    stride = stride*2;
  }

  stride = BLOCK_SIZE/2;
  while(stride > 0) {
    __syncthreads();
    int index = (threadIdx.x+1)*stride*2 - 1;
    if ((index+stride) < 2*BLOCK_SIZE)
      T[index+stride] += T[index];
    stride = stride / 2;
  }

  __syncthreads();
  if (start + t < len) {
      output[start + t] = T[t];
  }
  if (start + BLOCK_SIZE + t < len) {
      output[start + BLOCK_SIZE + t] = T[BLOCK_SIZE + t];
  }
  __syncthreads();

  if (t == BLOCK_SIZE - 1 && AuxSum != NULL) {
      AuxSum[blockIdx.x] = T[2 * BLOCK_SIZE - 1];
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  float *AuxSum;
  int numElements; // number of elements in the list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&AuxSum, ceil(numElements / (2.0 * BLOCK_SIZE)) * sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid((numElements - 1) / (BLOCK_SIZE * 2) + 1, 1, 1);
  dim3 DimGridAdd(1, 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numElements, AuxSum);
  scan<<<DimGridAdd, DimBlock>>>(AuxSum, AuxSum, ceil(numElements / (2.0 * BLOCK_SIZE)), NULL);
  add<<<DimGrid, DimBlock>>>(AuxSum, deviceOutput, numElements);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
