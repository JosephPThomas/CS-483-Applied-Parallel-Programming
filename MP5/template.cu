#include "hip/hip_runtime.h"
// MP Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void total(float *input, float *output, int len) {
  __shared__ float computedSum[2 * BLOCK_SIZE];
  unsigned int tx = threadIdx.x;
  unsigned int start = 2 * blockDim.x * blockIdx.x;
  if (start + tx < len)
    computedSum[tx] = input[start + tx];
  else
    computedSum[tx] = 0;
  if (start + blockDim.x + tx < len)
    computedSum[blockDim.x + tx] = input[start + blockDim.x + tx];
  else
    computedSum[blockDim.x + tx] = 0;
  for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) {
    __syncthreads();
    if (tx < stride)
      computedSum[tx] += computedSum[tx + stride];
  }
  if (tx == 0)
    output[blockIdx.x] = computedSum[0];
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  numOutputElements = numInputElements / (BLOCK_SIZE * 2);
  if (numInputElements % (BLOCK_SIZE * 2)) {
    numOutputElements++;
  }
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);
  wbLog(TRACE, "The number of output elements in the input is ",
        numOutputElements);

  int outputSize = numOutputElements * sizeof(float);
  int inputSize = numInputElements * sizeof(float);
  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput, inputSize);
  hipMalloc((void **)&deviceOutput, outputSize);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, inputSize, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");
  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid((numInputElements - 1) / BLOCK_SIZE + 1, 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, outputSize, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  /********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input. For simplicity, we do not
   * require that for this lab.
   ********************************************************************/
  for (ii = 1; ii < numOutputElements; ii++) {
    hostOutput[0] += hostOutput[ii];
  }

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceOutput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}
